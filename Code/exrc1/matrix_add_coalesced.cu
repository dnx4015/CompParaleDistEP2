
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

#define uint unsigned int
#define H2D hipMemcpyHostToDevice
#define D2H hipMemcpyDeviceToHost
#define DEBUG if(0)

#define REP(i,n) for(int i=0;i<n;i++)
#define FOR(i,a,b) for(int i=a;i<=b;i++)
#define INC(i,n,inc) for(int i=0;i<n;i+=inc)
#define imin(a,b) (a<b?a:b)

const int BlockSizeX = 32;
const int Factor = 4;
const int BlockSizeY = BlockSizeX/Factor;

inline hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG) 
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", 
			hipGetErrorString(result));
  }
#endif
  return result;
}

__global__ void add( int* matrix_result, 
                      int* matrix_a, 
                      int* matrix_b, 
                      const int N ) {
	int col = blockIdx.x * BlockSizeX + threadIdx.x;
    int row = blockIdx.y * BlockSizeX + threadIdx.y;
	
	INC(i, BlockSizeX, BlockSizeY){
		matrix_result[(row + i) * N + col] = 
			matrix_a[(row + i) * N + col] +
			matrix_b[(row + i) * N + col];
	}
}

int main(int argc, char* argv[]) {
	if (argc < 2) {
      fprintf(stderr, "Syntax: %s <vector size N>\n", argv[0]);
      return EXIT_FAILURE;
    }

    int N = atoi(argv[1]);

	if (N > 512) {
      fprintf(stderr, "Syntax: %s <vector size N (<=512)>\n", argv[0]);
      return EXIT_FAILURE;
    }

	hipEvent_t beginEvent ;
	hipEvent_t endEvent ;

	hipEventCreate( &beginEvent ) ;
	hipEventCreate( &endEvent ) ;

	hipEventRecord( beginEvent , 0 ) ;


	//const int N = 32;
	const int mSize = N*N*sizeof(int);
	int gridSize = N / BlockSizeX;

	int mul = 5;

	int host_a[N][N], host_b[N][N], host_result[N][N];
	int *dev_a, *dev_b, *dev_result;

 	REP(i, N){
		REP(j, N){
			host_a[i][j] = i * mul;
			host_b[i][j] = i;
		}
	}

    // allocate the memory on the GPU
    checkCuda( hipMalloc( (void**)&dev_a, mSize ) );
    checkCuda( hipMalloc( (void**)&dev_b, mSize ) );
    checkCuda( hipMalloc( (void**)&dev_result, mSize));

    // copy the arrays 'a' and 'b' to the GPU
    checkCuda(hipMemcpy(dev_a, host_a, mSize, H2D));
    checkCuda(hipMemcpy(dev_b, host_b, mSize, H2D)); 

	dim3 dimGrid(gridSize, gridSize, 1);
	dim3 dimBlock(BlockSizeX, BlockSizeX/Factor, 1);

    add<<<dimGrid,dimBlock>>>(dev_result, dev_a, dev_b, N);
	
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error!=hipSuccess) {
	   	fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
		exit(-1);
	}
    // copy the array 'result' back from the GPU to the CPU
    checkCuda( hipMemcpy( host_result, dev_result, mSize, D2H) );

	REP(i, N){
		REP(j, N){
			DEBUG printf("(%d) ", i*(mul+1));
			DEBUG printf("%d ", host_result[i][j]);
			assert(host_result[i][j] == i*(mul+1));	
		}	
		DEBUG printf("\n");
	}

    // free memory on the gpu side
    checkCuda( hipFree( dev_a ) );
    checkCuda( hipFree( dev_b ) );
    checkCuda( hipFree( dev_result ) );

	hipEventRecord( endEvent , 0 ) ;
	hipEventSynchronize( endEvent ) ;

	float timeValue ;
	hipEventElapsedTime( &timeValue , beginEvent , endEvent ) ;
	printf( "Time: %.2fs\n" , timeValue ) ;
	printf("Done\n");
}
