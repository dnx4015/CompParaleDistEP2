
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

#define uint unsigned int
#define H2D hipMemcpyHostToDevice
#define D2H hipMemcpyDeviceToHost
#define DEBUG if(0)

#define REP(i,n) for(int i=0;i<n;i++)
#define FOR(i,a,b) for(int i=a;i<=b;i++)
#define imin(a,b) (a<b?a:b)

#define BLOCK_SIZE 16

inline hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG) 
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", 
			hipGetErrorString(result));
  }
#endif
  return result;
}

__global__ void mult( int* matrix_result, 
                      int* matrix_a, 
                      int* matrix_b, 
                      int N ) {
    int val = 0;
	int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;

	REP(i, N){
		val += matrix_a[col * N + i] * matrix_b[i * N + row];
	}
	matrix_result[col * N + row] = val;
}

int main(int argc, char* argv[]) {
	if (argc < 2) {
      fprintf(stderr, "Syntax: %s <vector size N (<=512)>\n", argv[0]);
      return EXIT_FAILURE;
    }

    int N = atoi(argv[1]);

	if (N > 512) {
      fprintf(stderr, "Syntax: %s <vector size N (<=512)>\n", argv[0]);
      return EXIT_FAILURE;
    }

	hipEvent_t beginEvent ;
	hipEvent_t endEvent ;

	hipEventCreate( &beginEvent ) ;
	hipEventCreate( &endEvent ) ;

	hipEventRecord( beginEvent , 0 ) ;

	//const int N = 4;
	int mSize = N * N * sizeof(int);
	int col_sum = N * (N - 1) / 2;
	int mul = 5;

	int host_a[N][N], host_b[N][N], host_result[N][N];
	int *dev_a, *dev_b, *dev_result;

 	REP(i, N){
		REP(j, N){
			host_a[i][j] = i * mul;
			host_b[i][j] = i;
		}
	}

    // allocate the memory on the GPU
    checkCuda( hipMalloc( (void**)&dev_a, mSize ) );
    checkCuda( hipMalloc( (void**)&dev_b, mSize ) );
    checkCuda( hipMalloc( (void**)&dev_result, mSize));

    // copy the arrays 'a' and 'b' to the GPU
    checkCuda(hipMemcpy(dev_a, host_a, mSize, H2D));
    checkCuda(hipMemcpy(dev_b, host_b, mSize, H2D));

	int gridSize = N / BLOCK_SIZE;
	dim3 dimGrid(gridSize, gridSize, 1);
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

    mult<<<dimGrid,dimBlock>>>(dev_result, dev_a, dev_b, N );
	
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error!=hipSuccess) {
	   	fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
		exit(-1);
	}
    // copy the array 'result' back from the GPU to the CPU
    checkCuda( hipMemcpy( host_result, dev_result, mSize, D2H )); 

	REP(i, N){
		REP(j, N){
			DEBUG printf("(%d) ", i*mul*col_sum);
			DEBUG printf("%d ", host_result[i][j]);
			assert(host_result[i][j] == i*mul*col_sum);	
		}	
		DEBUG printf("\n");
	}

    // free memory on the gpu side
    checkCuda( hipFree( dev_a ) );
    checkCuda( hipFree( dev_b ) );
    checkCuda( hipFree( dev_result ) );

	hipEventRecord( endEvent , 0 ) ;
	hipEventSynchronize( endEvent ) ;

	float timeValue ;
	hipEventElapsedTime( &timeValue , beginEvent , endEvent ) ;
	printf( "Time: %.2fs\n" , timeValue ) ;

}
